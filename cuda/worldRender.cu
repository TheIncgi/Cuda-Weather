#include "hip/hip_runtime.h"
#ifdef __CDT_PARSER__
#define __global__
#define __device__
#define __shared__
#endif
#include "common.cu"
#include "vectors.cu"
#include "perlin.cu"

__device__ vec4 sunshineColorArgb( float lat, float lon, int* worldSize, float* worldTime ) {
	float s = sunshine(lat, lon, worldSize, worldTime);
	vec4 in;
	in.x = 35;//h
	in.y = 1-s*s; //s
	in.z = max(s, .3); //v
	in.w = 1; //a
	return hsvToArgb(in);
};
// |~~~~~~~~~~~~~~~~~~~~~
// |     Render code
// |~~~~~~~~~~~~~~~~~~~~~

__device__ float distanceToEdge(int a, int b, float af, float bf){
	float da = af - a;
	float db = bf - b;
	float thresh = .6;
	return min(1.0, max( 0.0, max( abs(thresh-da), abs(thresh-db) ) ));
}
__device__ int colorAt(int lat, int lon, int** groundType, float** groundMoistureIn){
	int fragColor = 0xFF000000;
	int gType = groundType[lat][lon];
	if(gType==ICE){
				fragColor = 0xFFFFFFFF;
			}else if(gType==SAND){
				fragColor = 0xFFFFEBCD;
			}else if(gType==DIRT){
				fragColor = 0xFFBC8F8F;
			}else if(gType==OCEAN){
				fragColor = 0xFF4169E1;
			}else if(gType==GRASS){
				fragColor = 0xFF90EE90;
			}else if(gType==STONE){
				fragColor = 0xFFD3D3D3;
			}else if(gType==FOREST){
				fragColor = 0xFF228B22;
			}else if(gType==LAKE){
				fragColor = 0xFF00FFFF;
			}

			vec4 hsvColor = argbToHsv(hexToArgb(fragColor));
			hsvColor.z *= map( clamp(groundMoistureIn[lat][lon],0,1), 0 ,1, 1, .75 );

			return fragColor;
}

//star background
__device__ int  backgroundColor(int n, int*imgSize){
	return 0;
}
__device__ void renderFlat(
		int i,
		int*     worldSize,
		//float*   worldSpeed,
		float**  elevation,
		int**    groundType,

		//inputs
		float*   worldTimeIn,

		float**  groundMoistureIn,
		float**  snowCoverIn,

		float*** temperatureIn,
		float*** pressureIn,
		float*** humidityIn,
		float*** cloudCoverIn,
		float*** windSpeedIn,

		int*   imageSize,
		int* imageOut){

		int x = i % imageSize[0];
		int y = i / imageSize[0];

//		int cx = imageSize[0]/2;
//		int cy = imageSize[1]/2;
		int lat    = y *         worldSize[0]  / imageSize[1];
		int lon    = x *         worldSize[1]  / imageSize[0];
		float latf = y * ((float)worldSize[0]) / imageSize[1] + .5;
		float lonf = x * ((float)worldSize[1]) / imageSize[0] + .5;

		float tileWidth  = imageSize[0] / (float)worldSize[1];
		float tileHeight = imageSize[1] / (float)worldSize[0];

		float offX = perlin(latf, lonf, .25, 999, 45, worldSize);
		float offY = perlin(latf, lonf, .25, 27, 98, worldSize);

		int theColor = colorAt(lat, lon, groundType, groundMoistureIn);
		int blendColor = colorAt(
				((int)(latf + offX)) % worldSize[0],
				((int)(lonf + offY)) % worldSize[1],
				groundType,
				groundMoistureIn
		);
		offX = perlin(latf, lonf, .1, 7798, 45, worldSize);
		offY = perlin(latf, lonf, .1, 991, 98, worldSize);

		blendColor = mixColors_hex(blendColor, colorAt(
				((int)(latf + offX)) % worldSize[0],
				((int)(lonf + offY)) % worldSize[1],
				groundType,
				groundMoistureIn), .5);

		int PERL = ((int)( (offX+1)*127 )) + 0xFF000000 ;

		vec4 sunColor = sunshineColorArgb(latf, lonf, worldSize, worldTimeIn);
		theColor = argbToHex(multipyColor(hexToArgb(theColor), sunColor));
		imageOut[i] = theColor;//mixColors(theColor, blendColor, .5*distanceToEdge(lat, lon, latf, lonf));
//		if(lat == 47)
//			fragColor = 0xFFFF00FF;
//		fragColor = 0xFF000000 | ((int)(lat/((float)worldSize[0])*255));
//		fragColor |= ((int)(lon/((float)worldSize[1])*255)) << 8;

//		if(lat < 0 )
//			fragColor = 0xFFFF0000;
//		if(lat >= worldSize[0])
//			fragColor = 0xFFFF0000;





		//imageOut[i] = fragColor;//, 0, 0xFFFFFFFF);
}

extern "C"
__global__ void render(
		//static
			int*     worldSize,
			//float*   worldSpeed,
			float**  elevation,
			int**    groundType,

			//inputs
			float*   worldTimeIn,

			float**  groundMoistureIn,
			float**  snowCoverIn,

			float*** temperatureIn,
			float*** pressureIn,
			float*** humidityIn,
			float*** cloudCoverIn,
			float*** windSpeedIn,

			int*   imageSize,
			int* imageOut
			) {
	int i = getGlobalThreadID();
	if(i>= imageSize[0]*imageSize[1]) return;

	renderFlat(i, worldSize, elevation, groundType, worldTimeIn, groundMoistureIn, snowCoverIn, temperatureIn, pressureIn, humidityIn, cloudCoverIn, windSpeedIn, imageSize, imageOut);

}
