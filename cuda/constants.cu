#pragma once
#include <hip/hip_runtime.h>

#ifdef __CDT_PARSER__
#define __global__
#define __device__
#define __shared__
#endif

float __constant__ PI     = 3.14159654;
float __constant__ E      = 2.71828182846;
float __constant__ HALF_C = 3.14159654/180;
float __constant__ sqrt2  = 1.41421356237;
float __constant__ sqrt3  = 1.73205080757;

float __constant__ PLANET_RADIUS = 6371; //km
float __constant__ PLANET_MASS   = 5.972e24; //kg ...that's 5 septillion
float __constant__ PLANET_TILT   = 23.5;
float __constant__ GRAVITATIONAL_CONSTANT = 6.67E-11;
float __constant__ STEFAN_BOLTZMANN_CONSTANT = 5.670374E-8;

int __constant__ SQ_M_IN_SQ_KM = 1000000;

int __constant__ SAND     = 0;
int __constant__ DIRT     = 1;
int __constant__ OCEAN    = 2;
int __constant__ GRASS    = 3; //promoted from dirt in low rain, dry climates
int __constant__ STONE    = 4;
int __constant__ ICE      = 5; //ocean, but past 75 degrees, arctic circles are about 66, but that's seasonally related
int __constant__ FOREST   = 6; //promoted from dirt, humid climates
int __constant__ LAKE     = 7; //local minima of rainy areas

__device__ const char * biomeName(int groundType) {
	switch(groundType) {
		case 0: return "SAND";
		case 1: return "DIRT";
		case 2: return "OCEAN";
		case 3: return "GRASS";
		case 4: return "STONE";
		case 5: return "ICE";
		case 6: return "FOREST";
		case 7: return "LAKE";
		default: return "???";
	}
}